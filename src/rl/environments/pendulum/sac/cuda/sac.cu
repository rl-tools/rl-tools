#include "hip/hip_runtime.h"
// /usr/local/cuda/bin/nvcc -I include -DRL_TOOLS_BACKEND_ENABLE_CUDA -lcublas src/rl/environments/pendulum/sac/cuda/sac.cu

#ifdef RL_TOOLS_DEBUG
#define RL_TOOLS_DEBUG_DEVICE_CUDA_SYNCHRONIZE_STATUS_CHECK
#endif
#define RL_TOOLS_OPERATIONS_CPU_MUX_INCLUDE_CUDA
#include <rl_tools/operations/cpu_mux.h>
#include <rl_tools/nn/optimizers/adam/instance/operations_cuda.h>
#include <rl_tools/nn/operations_cpu_mux.h>
#include <rl_tools/nn/loss_functions/mse/operations_cuda.h>
#include <rl_tools/nn/layers/sample_and_squash/operations_cuda.h>
#include <rl_tools/rl/environments/pendulum/operations_cpu.h>
#include <rl_tools/nn_models/mlp/operations_generic.h>
#include <rl_tools/nn_models/random_uniform/operations_generic.h>
#include <rl_tools/nn_models/sequential/operations_generic.h>

#include <rl_tools/nn/optimizers/adam/operations_cuda.h>

#include <rl_tools/rl/components/off_policy_runner/operations_cuda.h>

#include <rl_tools/rl/algorithms/sac/operations_cuda.h>
#include <rl_tools/rl/algorithms/sac/loop/core/config.h>
#include <rl_tools/rl/loop/steps/evaluation/config.h>
#include <rl_tools/rl/loop/steps/timing/config.h>
#include <rl_tools/rl/algorithms/sac/loop/core/operations_generic.h>
#include <rl_tools/rl/loop/steps/evaluation/operations_generic.h>
#include <rl_tools/rl/loop/steps/timing/operations_cpu.h>

#include "cuda_graph_export.h"

namespace rlt = rl_tools;

using DEVICE = rlt::devices::DEVICE_FACTORY_CUDA<>;
#ifndef _MSC_VER
using DEVICE_EVALUATION = rlt::devices::DEVICE_FACTORY<>;
#else
using DEVICE_INIT = rlt::devices::DefaultCPU; // for some reason MKL makes problems in this case (this example seems cursed)
#endif
DEVICE dummy_device; // this is needed because default_engine can not take a const device
using TI = typename DEVICE::index_t;
// using T = float;
using TYPE_POLICY = rlt::numeric_types::Policy<float>;
using RNG = DEVICE::SPEC::RANDOM::ENGINE<rlt::devices::random::CUDA::Specification<TI, 1024>>;


using PENDULUM_SPEC = rlt::rl::environments::pendulum::Specification<float, TI, rlt::rl::environments::pendulum::DefaultParameters<float>>;
using ENVIRONMENT = rlt::rl::environments::Pendulum<PENDULUM_SPEC>;
struct LOOP_CORE_PARAMETERS: rlt::rl::algorithms::sac::loop::core::DefaultParameters<TYPE_POLICY, TI, ENVIRONMENT>{
    struct SAC_PARAMETERS: rlt::rl::algorithms::sac::DefaultParameters<TYPE_POLICY, TI, ENVIRONMENT::ACTION_DIM>{
        static constexpr TI ACTOR_BATCH_SIZE = 100;
        static constexpr TI CRITIC_BATCH_SIZE = 100;
    };
    static constexpr TI STEP_LIMIT = 10000;
    static constexpr TI REPLAY_BUFFER_CAP = STEP_LIMIT;
    static constexpr TI ACTOR_NUM_LAYERS = 3;
    static constexpr TI ACTOR_HIDDEN_DIM = 64;
    static constexpr TI CRITIC_NUM_LAYERS = 3;
    static constexpr TI CRITIC_HIDDEN_DIM = 64;
    static constexpr bool COLLECT_EPISODE_STATS = false;
    static constexpr TI EPISODE_STATS_BUFFER_SIZE = 0;
};
template <typename RNG>
using LOOP_CORE_CONFIG = rlt::rl::algorithms::sac::loop::core::Config<TYPE_POLICY, TI, RNG, ENVIRONMENT, LOOP_CORE_PARAMETERS>;

struct LOOP_EVAL_PARAMETERS: rlt::rl::loop::steps::evaluation::Parameters<TYPE_POLICY, TI, LOOP_CORE_CONFIG<RNG>>{
    static constexpr TI NUM_EVALUATION_EPISODES = 100;
};
template <typename RNG>
using LOOP_EVAL_CONFIG = rlt::rl::loop::steps::evaluation::Config<LOOP_CORE_CONFIG<RNG>, LOOP_EVAL_PARAMETERS>;
template <typename RNG>
using LOOP_CONFIG = LOOP_EVAL_CONFIG<RNG>;

using LOOP_STATE = typename LOOP_CONFIG<RNG>::template State<LOOP_CONFIG<RNG>>;


int main() {
    TI seed = 0;
    DEVICE device;
    DEVICE_EVALUATION device_evaluation;
    LOOP_STATE ts;
    using CONFIG = typename decltype(ts)::CONFIG;
    using CORE_PARAMETERS = typename CONFIG::CORE_PARAMETERS;
    using EVAL_PARAMETERS = typename CONFIG::EVALUATION_PARAMETERS;
    DEVICE_EVALUATION::SPEC::RANDOM::ENGINE rng_evaluation;
    rlt::init(device_evaluation, rng_evaluation, seed);
    using ACTOR_TYPE_ORIG = rlt::utils::typing::remove_reference_t<decltype(rlt::get_actor(ts))>;
    using ACTOR_TYPE_INFERENCE = ACTOR_TYPE_ORIG:: template CHANGE_CAPABILITY<rlt::nn::capability::Forward<>>;
    using ACTOR_TYPE_EVALUATION = ACTOR_TYPE_INFERENCE:: template CHANGE_BATCH_SIZE<TI, EVAL_PARAMETERS::NUM_EVALUATION_EPISODES>;
    ACTOR_TYPE_EVALUATION actor_evaluation;
    ACTOR_TYPE_EVALUATION::Buffer<> actor_buffers_evaluation;
    ENVIRONMENT env_evaluation;
    ENVIRONMENT::Parameters env_evaluation_parameters;
    rlt::rl::environments::DummyUI ui;
    rlt::init(device);
    rlt::malloc(device, ts);
    rlt::malloc(device_evaluation, actor_evaluation);
    rlt::malloc(device_evaluation, actor_buffers_evaluation);
    rlt::init(device, ts, 1);
    TI step = 0;
    bool finished = false;

    // {
    //     hipGraph_t test_graph;
    //     hipStreamBeginCapture(device.stream, hipStreamCaptureModeGlobal);
    //     device.graph_capture_active = true;
    //     rlt::zero_gradient(device, ts.actor_critic.actor.content.input_layer);
    //     hipStreamEndCapture(device.stream, &test_graph);
    //     device.graph_capture_active = false;
    //
    //     rlt::check_status(device);
    //     hipGraphExec_t graphExec;
    //     hipGraphInstantiate(&graphExec, test_graph, nullptr, nullptr, 0);
    //     hipGraphLaunch(graphExec, device.stream);
    //     hipDeviceSynchronize();
    // }

    constexpr bool CUDA_GRAPH = true;
    // constexpr bool CUDA_GRAPH = false;



    if constexpr(CUDA_GRAPH){
        hipGraph_t step_graph;
        hipGraphExec_t step_graph_exec;
        {
            hipStreamBeginCapture(device.stream, hipStreamCaptureModeGlobal);
            device.graph_capture_active = true;
            rlt::step<1>(device, ts.off_policy_runner, ts.actor_critic.actor, ts.actor_buffers_eval, ts.rng);
            hipStreamEndCapture(device.stream, &step_graph);
            device.graph_capture_active = false;
            rlt::check_status(device);
            hipGraphInstantiate(&step_graph_exec, step_graph, nullptr, nullptr, 0);
        }
        hipGraph_t critic_training_graph;
        hipGraphExec_t critic_training_graph_exec;
        {
            hipStreamBeginCapture(device.stream, hipStreamCaptureModeGlobal);
            device.graph_capture_active = true;
            for(int critic_i = 0; critic_i < 2; critic_i++){
                rlt::gather_batch(device, ts.off_policy_runner, ts.critic_batch, ts.rng);
                rlt::randn(device, ts.action_noise_critic, ts.rng);
                rlt::train_critic(device, ts.actor_critic, ts.actor_critic.critics[critic_i], ts.critic_batch, ts.actor_critic.critic_optimizers[critic_i], ts.actor_target_buffers[critic_i], ts.critic_buffers[critic_i], ts.critic_target_buffers[critic_i], ts.critic_training_buffers[critic_i], ts.action_noise_critic, ts.rng);
            }
            hipStreamEndCapture(device.stream, &critic_training_graph);
            device.graph_capture_active = false;
            rlt::print_graph(critic_training_graph);
            rlt::check_status(device);
            hipGraphInstantiate(&critic_training_graph_exec, critic_training_graph, nullptr, nullptr, 0);
        }
        hipGraph_t actor_training_graph;
        hipGraphExec_t actor_training_graph_exec;
        {
            hipStreamBeginCapture(device.stream, hipStreamCaptureModeGlobal);
            device.graph_capture_active = true;
            rlt::gather_batch(device, ts.off_policy_runner, ts.actor_batch, ts.rng);
            rlt::randn(device, ts.action_noise_actor, ts.rng);
            rlt::train_actor(device, ts.actor_critic, ts.actor_batch, ts.actor_critic.actor_optimizer, ts.actor_buffers[0], ts.critic_buffers[0], ts.actor_training_buffers, ts.action_noise_actor, ts.rng);
            rlt::update_critic_targets(device, ts.actor_critic);
            hipStreamEndCapture(device.stream, &actor_training_graph);
            device.graph_capture_active = false;
            rlt::check_status(device);
            hipGraphInstantiate(&actor_training_graph_exec, actor_training_graph, nullptr, nullptr, 0);
            rlt::print_graph(actor_training_graph);
        }
        dumpCudaGraphDOT(step_graph,            "cuda_graph_step.dot");
        dumpCudaGraphDOT(critic_training_graph, "cuda_graph_critic_training.dot");
        dumpCudaGraphDOT(actor_training_graph,  "cuda_traph_actor_training.dot");
        while(!finished){
            // Evaluation
            if(step % 1000 == 0){
                rlt::copy(device, device_evaluation, rlt::get_actor(ts), actor_evaluation);
                hipStreamSynchronize(device.stream);
                using RESULT_SPEC = rlt::rl::utils::evaluation::Specification<TYPE_POLICY, TI, typename decltype(ts)::CONFIG::ENVIRONMENT_EVALUATION, EVAL_PARAMETERS::NUM_EVALUATION_EPISODES, CORE_PARAMETERS::EPISODE_STEP_LIMIT>;
                rlt::rl::utils::evaluation::Result<RESULT_SPEC> result;
                rlt::evaluate(device_evaluation, env_evaluation, ui, actor_evaluation, result, rng_evaluation, rlt::Mode<rlt::mode::Evaluation<>>{});
                rlt::log(device_evaluation, device_evaluation.logger, "Step: ", step, " Mean return: ", result.returns_mean);
            }

            rlt::set_step(device, device.logger, step);
            hipGraphLaunch(step_graph_exec, device.stream);
            rlt::check_status(device);
            if(step > CONFIG::CORE_PARAMETERS::N_WARMUP_STEPS){
                if(step % CONFIG::CORE_PARAMETERS::SAC_PARAMETERS::CRITIC_TRAINING_INTERVAL == 0) {
                    hipGraphLaunch(critic_training_graph_exec, device.stream);
                    rlt::check_status(device);
                }
                if(step % CONFIG::CORE_PARAMETERS::SAC_PARAMETERS::ACTOR_TRAINING_INTERVAL == 0) {
                    hipGraphLaunch(actor_training_graph_exec, device.stream);
                    rlt::check_status(device);
                }
            }
            step++;
            finished = step > CORE_PARAMETERS::STEP_LIMIT;
         }
    }
    else {
        while(!finished){
            // Evaluation
            if(step % 1000 == 0){
                rlt::copy(device, device_evaluation, rlt::get_actor(ts), actor_evaluation);
                hipStreamSynchronize(device.stream);
                using RESULT_SPEC = rlt::rl::utils::evaluation::Specification<TYPE_POLICY, TI, typename decltype(ts)::CONFIG::ENVIRONMENT_EVALUATION, EVAL_PARAMETERS::NUM_EVALUATION_EPISODES, CORE_PARAMETERS::EPISODE_STEP_LIMIT>;
                rlt::rl::utils::evaluation::Result<RESULT_SPEC> result;
                rlt::evaluate(device_evaluation, env_evaluation, ui, actor_evaluation, result, rng_evaluation, rlt::Mode<rlt::mode::Evaluation<>>{});
                rlt::log(device_evaluation, device_evaluation.logger, "Step: ", step, " Mean return: ", result.returns_mean);
            }

            // Training
            rlt::set_step(device, device.logger, step);
            rlt::step<1>(device, ts.off_policy_runner, ts.actor_critic.actor, ts.actor_buffers_eval, ts.rng);
            if(step > CONFIG::CORE_PARAMETERS::N_WARMUP_STEPS){
                if(step % CONFIG::CORE_PARAMETERS::SAC_PARAMETERS::CRITIC_TRAINING_INTERVAL == 0) {
                    for(TI critic_i = 0; critic_i < 2; critic_i++){
                        rlt::gather_batch(device, ts.off_policy_runner, ts.critic_batch, ts.rng);
                        rlt::randn(device, ts.action_noise_critic, ts.rng);
                        rlt::train_critic(device, ts.actor_critic, ts.actor_critic.critics[critic_i], ts.critic_batch, ts.actor_critic.critic_optimizers[critic_i], ts.actor_target_buffers[critic_i], ts.critic_buffers[critic_i], ts.critic_target_buffers[critic_i], ts.critic_training_buffers[critic_i], ts.action_noise_critic, ts.rng);
                    }
                }
                if(step % CONFIG::CORE_PARAMETERS::SAC_PARAMETERS::ACTOR_TRAINING_INTERVAL == 0) {
                    rlt::gather_batch(device, ts.off_policy_runner, ts.actor_batch, ts.rng);
                    rlt::randn(device, ts.action_noise_actor, ts.rng);
                    rlt::train_actor(device, ts.actor_critic, ts.actor_batch, ts.actor_critic.actor_optimizer, ts.actor_buffers[0], ts.critic_buffers[0], ts.actor_training_buffers, ts.action_noise_actor, ts.rng);
                    rlt::update_critic_targets(device, ts.actor_critic);
                }
            }
            step++;
            finished = step > CORE_PARAMETERS::STEP_LIMIT;
         }

    }
    rlt::free(device, ts);
    return 0;
}

// benchmark training should take < 2s on P1
