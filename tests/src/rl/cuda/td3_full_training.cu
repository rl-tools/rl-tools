// ------------ Groups 1 ------------
#include <rl_tools/operations/cuda/group_1.h>
#include <rl_tools/operations/cpu_mkl/group_1.h>
#include <rl_tools/operations/cpu_tensorboard/group_1.h>
// ------------ Groups 2 ------------
#include <rl_tools/operations/cuda/group_2.h>
#include <rl_tools/operations/cpu_mkl/group_2.h>
#include <rl_tools/operations/cpu_tensorboard/group_2.h>
// ------------ Groups 3 ------------
#include <rl_tools/operations/cuda/group_3.h>
#include <rl_tools/operations/cpu_mkl/group_3.h>
#include <rl_tools/operations/cpu_tensorboard/group_3.h>

namespace rlt = RL_TOOLS_NAMESPACE_WRAPPER ::rl_tools;

#include <rl_tools/nn/optimizers/adam/instance/operations_cuda.h>
#include <rl_tools/nn/operations_cuda.h>
#include <rl_tools/nn/operations_cpu_mkl.h>
using DEV_SPEC_INIT = rlt::devices::cpu::Specification<rlt::devices::math::CPU, rlt::devices::random::CPU, rlt::devices::logging::CPU_TENSORBOARD<>>;
using DEVICE_INIT = rlt::devices::CPU<DEV_SPEC_INIT>;
//using DEVICE = rlt::devices::CPU_MKL<DEV_SPEC_INIT>;
using DEVICE = rlt::devices::DefaultCUDA;
using TI = DEVICE::index_t;
using DEV_SPEC = DEVICE::SPEC;

#include "td3_full_training_parameters_pendulum.h"

#include <rl_tools/nn_models/operations_generic.h>
#include <rl_tools/nn/optimizers/adam/operations_generic.h>
#include <rl_tools/rl/components/off_policy_runner/operations_cuda.h>
#include <rl_tools/rl/algorithms/td3/operations_cuda.h>
#include <rl_tools/rl/algorithms/td3/operations_generic.h>

#include <rl_tools/rl/utils/evaluation/operations_generic.h>


#include <gtest/gtest.h>
#include <filesystem>

using T = float;


using p = parameters_pendulum_0<DEVICE, T>;
using rlp = p::rl<p::env::ENVIRONMENT>;

static_assert(rlp::ACTOR_CRITIC_TYPE::SPEC::PARAMETERS::ACTOR_BATCH_SIZE == rlp::ACTOR_CRITIC_TYPE::SPEC::PARAMETERS::CRITIC_BATCH_SIZE);

TEST(RL_TOOLS_RL_CUDA_TD3, TEST_FULL_TRAINING) {
    DEVICE_INIT::SPEC::LOGGING logger;
    DEVICE device;
    DEVICE_INIT device_init;
    rlp::OPTIMIZER optimizer;

    rlp::ACTOR_CRITIC_TYPE actor_critic_init;
    rlp::ACTOR_CRITIC_TYPE actor_critic;
    rlp::OFF_POLICY_RUNNER_TYPE off_policy_runner_init, off_policy_runner;
    rlp::OFF_POLICY_RUNNER_TYPE* off_policy_runner_pointer;

    rlp::CRITIC_BATCH_TYPE critic_batch;
    rlp::CRITIC_BATCH_TYPE* critic_batch_pointer;
    rlp::CRITIC_TRAINING_BUFFERS_TYPE critic_training_buffers;
    rlp::CRITIC_NETWORK_TYPE::Buffer<rlp::ACTOR_CRITIC_TYPE::SPEC::PARAMETERS::CRITIC_BATCH_SIZE> critic_buffers[2];

    rlp::ACTOR_BATCH_TYPE actor_batch;
    rlp::ACTOR_BATCH_TYPE* actor_batch_pointer;
    rlp::ACTOR_TRAINING_BUFFERS_TYPE actor_training_buffers;
    rlp::ACTOR_NETWORK_TYPE::Buffer<rlp::ACTOR_CRITIC_TYPE::SPEC::PARAMETERS::ACTOR_BATCH_SIZE> actor_buffers[2];
    rlp::ACTOR_NETWORK_TYPE::Buffer<rlp::OFF_POLICY_RUNNER_SPEC::PARAMETERS::N_ENVIRONMENTS> actor_buffers_eval;
    rlp::ACTOR_NETWORK_TYPE::Buffer<rlp::OFF_POLICY_RUNNER_SPEC::PARAMETERS::N_ENVIRONMENTS> actor_buffers_eval_init;

    rlt::init(device);
    rlt::construct(device_init, device_init.logger);
    auto rng_init = rlt::random::default_engine(DEVICE_INIT::SPEC::RANDOM());
    DEVICE::SPEC::RANDOM::ENGINE<> rng;
    rlt::malloc(device, rng);
    rlt::init(device, rng, 1);
    p::env::ENVIRONMENT envs[decltype(off_policy_runner_init)::N_ENVIRONMENTS];
    p::env::ENVIRONMENT::Parameters env_parameters[decltype(off_policy_runner_init)::N_ENVIRONMENTS];
    rlt::rl::environments::DummyUI ui;
    
    rlt::malloc(device, optimizer);
    rlt::init(device, optimizer);
    rlt::get_ref(device, optimizer.parameters, 0).epsilon_sqrt = 0;
    
    rlt::malloc(device_init, actor_critic_init);
    rlt::malloc(device, actor_critic);
    rlt::malloc(device_init, off_policy_runner_init);
    rlt::malloc(device, off_policy_runner);
    hipMalloc(&off_policy_runner_pointer, sizeof(rlp::OFF_POLICY_RUNNER_TYPE));
    rlt::check_status(device);

    rlt::malloc(device, critic_batch);
    hipMalloc(&critic_batch_pointer, sizeof(rlp::CRITIC_BATCH_TYPE));
    rlt::check_status(device);
    rlt::malloc(device, critic_training_buffers);
    rlt::malloc(device, critic_buffers[0]);
    rlt::malloc(device, critic_buffers[1]);

    rlt::malloc(device, actor_batch);
    hipMalloc(&actor_batch_pointer, sizeof(rlp::ACTOR_BATCH_TYPE));
    rlt::check_status(device);
    rlt::malloc(device, actor_training_buffers);
    rlt::malloc(device, actor_buffers_eval);
    rlt::malloc(device_init, actor_buffers_eval_init);
    rlt::malloc(device, actor_buffers[0]);
    rlt::malloc(device, actor_buffers[1]);

    rlt::init(device_init, actor_critic_init, rng_init);
    rlt::copy(device_init, device, actor_critic_init, actor_critic);
//    for(int i = 0; i < decltype(off_policy_runner_init)::N_ENVIRONMENTS; i += 1){
//        auto parameters = p::env::parameters;
//        envs[i].parameters = parameters;
//    }
    rlt::init(device_init, off_policy_runner_init, envs, env_parameters);
    rlt::copy(device_init, device, off_policy_runner_init, off_policy_runner);
    hipMemcpy(off_policy_runner_pointer, &off_policy_runner, sizeof(rlp::OFF_POLICY_RUNNER_TYPE), hipMemcpyHostToDevice);
    rlt::check_status(device);
    hipMemcpy(actor_batch_pointer, &actor_batch, sizeof(rlp::ACTOR_BATCH_TYPE), hipMemcpyHostToDevice);
    rlt::check_status(device);
    hipMemcpy(critic_batch_pointer, &critic_batch, sizeof(rlp::CRITIC_BATCH_TYPE), hipMemcpyHostToDevice);
    rlt::check_status(device);

    auto start_time = std::chrono::high_resolution_clock::now();

    constexpr DEVICE::index_t step_limit = 20000;
    for(int step_i = 0; step_i < step_limit; step_i += 1){
        rng = rlt::random::next(DEVICE::SPEC::RANDOM(), rng);
        rlt::rl::components::off_policy_runner::prologue(device, off_policy_runner_pointer, rng);
        rng = rlt::random::next(DEVICE::SPEC::RANDOM(), rng);
        rlt::rl::components::off_policy_runner::interlude(device, off_policy_runner, actor_critic.actor, actor_buffers_eval, rng);
        rng = rlt::random::next(DEVICE::SPEC::RANDOM(), rng);
        rlt::rl::components::off_policy_runner::epilogue(device, off_policy_runner_pointer, actor_critic.actor, rng);

        if(step_i % 1000 == 0){
            auto current_time = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> elapsed_seconds = current_time - start_time;
            std::cout << "step_i: " << step_i << " " << elapsed_seconds.count() << "s" << std::endl;
        }

        if(step_i > rlp::ACTOR_CRITIC_PARAMETERS::N_WARMUP_STEPS_CRITIC && step_i % rlp::ACTOR_CRITIC_PARAMETERS::CRITIC_TRAINING_INTERVAL == 0) {
            for (int critic_i = 0; critic_i < 2; critic_i++) {
                rng = rlt::random::next(DEVICE::SPEC::RANDOM(), rng);
//                hipDeviceSynchronize();
//                auto start = std::chrono::high_resolution_clock::now();
                rlt::target_action_noise(device, actor_critic, critic_training_buffers.target_next_action_noise, rng);
                rng = rlt::random::next(DEVICE::SPEC::RANDOM(), rng);
                rlt::gather_batch(device, off_policy_runner_pointer, critic_batch, rng);
                rlt::train_critic(device, actor_critic, critic_i == 0 ? actor_critic.critic_1 : actor_critic.critic_2, critic_batch, optimizer, actor_buffers[critic_i], critic_buffers[critic_i], critic_training_buffers, rng);
//                hipDeviceSynchronize();
//                auto end = std::chrono::high_resolution_clock::now();
//                auto duration_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
//                std::cout << "critic_i: " << critic_i << " " << duration_microseconds << "us" << std::endl;
            }
        }

        if(step_i > rlp::ACTOR_CRITIC_PARAMETERS::N_WARMUP_STEPS_ACTOR && step_i % rlp::ACTOR_CRITIC_PARAMETERS::ACTOR_TRAINING_INTERVAL == 0) {
            hipDeviceSynchronize();
//            auto start = std::chrono::high_resolution_clock::now();
            rng = rlt::random::next(DEVICE::SPEC::RANDOM(), rng);
            rlt::gather_batch(device, off_policy_runner_pointer, actor_batch, rng);
            rlt::train_actor(device, actor_critic, actor_batch, optimizer, actor_buffers[0], critic_buffers[0], actor_training_buffers, rng);
//            hipDeviceSynchronize();
//            auto end = std::chrono::high_resolution_clock::now();
//            auto duration_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
//                    std::cout << "actor: " << duration_microseconds << "us" << std::endl;
        }

        if(step_i > rlp::ACTOR_CRITIC_PARAMETERS::N_WARMUP_STEPS_CRITIC && step_i % rlp::ACTOR_CRITIC_PARAMETERS::CRITIC_TARGET_UPDATE_INTERVAL == 0) {
            {
//                hipDeviceSynchronize();
//                auto start = std::chrono::high_resolution_clock::now();
                rlt::update_critic_targets(device, actor_critic);
//                hipDeviceSynchronize();
//                auto end = std::chrono::high_resolution_clock::now();
//                auto duration_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
//                    std::cout << "update: " << duration_microseconds << "us" << std::endl;
            }
        }
        if(step_i > rlp::ACTOR_CRITIC_PARAMETERS::N_WARMUP_STEPS_ACTOR && step_i % rlp::ACTOR_CRITIC_PARAMETERS::ACTOR_TARGET_UPDATE_INTERVAL == 0) {
            {
//                hipDeviceSynchronize();
//                auto start = std::chrono::high_resolution_clock::now();
                rlt::update_actor_target(device, actor_critic);
//                hipDeviceSynchronize();
//                auto end = std::chrono::high_resolution_clock::now();
//                auto duration_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
//                    std::cout << "update: " << duration_microseconds << "us" << std::endl;
            }
        }
        if(step_i % 1000 == 0){
            rlt::copy(device, device_init, actor_critic, actor_critic_init);
            using RESULT_SPEC = rlt::rl::utils::evaluation::Specification<T, TI, p::env::ENVIRONMENT, 1, rlp::OFF_POLICY_RUNNER_PARAMETERS::EPISODE_STEP_LIMIT>;
            rlt::rl::utils::evaluation::Result<RESULT_SPEC> results;
            rlt::evaluate(device_init, envs[0], ui, actor_critic_init.actor, results, actor_buffers_eval_init, rng_init, false);
            std::cout << "Mean return: " << results.returns_mean << std::endl;
        }
    }
    {
        auto current_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = current_time - start_time;
        std::cout << "total time: " << elapsed_seconds.count() << "s" << std::endl;
        // 90s, 15x of CPU BLAS => todo: investigate individual kernel timings
        // on device rollout: 24s, 6x of CPU BLAS => todo: investigate individual kernel timings
        // no device sync: 14s, 2.5x of CPU BLAS => todo: investigate individual kernel timings

    }
    rlt::free(device, critic_batch);
    rlt::free(device, critic_training_buffers);
    rlt::free(device, actor_batch);
    rlt::free(device, actor_training_buffers);
    rlt::free(device, off_policy_runner);
    rlt::free(device, actor_critic);
}
